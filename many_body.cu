#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

// Kernel
__global__ void updateKernel(double *xPos, double *yPos, double *xVelo, double *yVelo, double *m, double *xPos2, double *yPos2, double *xVelo2, double *yVelo2, int n, double dt) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate the global thread index
    if (idx < n) { // Ensure we don't access out-of-bounds memory
        double xAccel = 0.0;
        double yAccel = 0.0;
        for (int i = 0; i < n; i++) {
            double xDist = xPos[i] - xPos[idx];
            double yDist = yPos[i] - yPos[idx];
            double distSqr = pow(xDist, 2.0) + pow(yDist, 2.0);
            xAccel = xAccel + (m[i] * xDist) / (distSqr + 0.5);
            yAccel = yAccel + (m[i] * yDist) / (distSqr + 0.5);
        }
        xVelo2[idx] = xVelo[idx] + xAccel * 6.6743e-11 * dt;
        yVelo2[idx] = yVelo[idx] + yAccel * 6.6743e-11 * dt;
        xPos2[idx] = xPos[idx] + xVelo2[idx] * dt;
        yPos2[idx] = yPos[idx] + yVelo2[idx] * dt;
    }
}

struct cParticleArray {
    double *xPos, *yPos, *xVelo, *yVelo, *mass;
    int n;
};

// Define 
int n;
double *dXPos, *dYPos, *dXVelo, *dYVelo, *dM, *dXPos2, *dYPos2, *dXVelo2, *dYVelo2;

extern "C" __declspec(dllexport) void freeArray(cParticleArray* ptr) {
    if (ptr) {
        // Free the dynamically allocated arrays
        delete[] ptr->xPos;
        delete[] ptr->yPos;
        delete[] ptr->xVelo;
        delete[] ptr->yVelo;
        // Free the struct itself
        delete ptr;
    }
}

extern "C" __declspec(dllexport) void prepare(double* xPos, double* yPos, double* xVelo, double* yVelo, double* mass, int counts) {

    n = counts;

    // Allocate device memory and perform computations...
    hipMalloc((void **)&dXPos, n * sizeof(double));
    hipMalloc((void **)&dYPos, n * sizeof(double));
    hipMalloc((void **)&dXVelo, n * sizeof(double));
    hipMalloc((void **)&dYVelo, n * sizeof(double));
    hipMalloc((void **)&dM, n * sizeof(double));
    hipMalloc((void **)&dXPos2, n * sizeof(double));
    hipMalloc((void **)&dYPos2, n * sizeof(double));
    hipMalloc((void **)&dXVelo2, n * sizeof(double));
    hipMalloc((void **)&dYVelo2, n * sizeof(double));

    hipMemcpyAsync(dXPos, xPos, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(dYPos, yPos, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(dXVelo, xVelo, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(dYVelo, yVelo, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(dM, mass, n * sizeof(double), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    updateKernel<<<blocksPerGrid, threadsPerBlock>>>(dXPos, dYPos, dXVelo, dYVelo, dM, dXPos2, dYPos2, dXVelo2, dYVelo2, n, 0);
    hipDeviceSynchronize();
}

extern "C" __declspec(dllexport) cParticleArray* update(double dt) {

    cParticleArray *particles = new cParticleArray;
    particles->xPos = new double[n];
    particles->yPos = new double[n];
    particles->xVelo = new double[n];
    particles->yVelo = new double[n];

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    updateKernel<<<blocksPerGrid, threadsPerBlock>>>(dXPos, dYPos, dXVelo, dYVelo, dM, dXPos2, dYPos2, dXVelo2, dYVelo2, n, dt);
    hipDeviceSynchronize();

    // Copy results internally in GPU
    hipMemcpyAsync(dXPos, dXPos2, n * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpyAsync(dYPos, dYPos2, n * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpyAsync(dXVelo, dXVelo2, n * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpyAsync(dYVelo, dYVelo2, n * sizeof(double), hipMemcpyDeviceToDevice);

    hipDeviceSynchronize();

    // Copy results back to CPU
    hipMemcpy(particles->xPos, dXPos, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(particles->yPos, dYPos, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(particles->xVelo, dXVelo, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(particles->yVelo, dYVelo, n * sizeof(double), hipMemcpyDeviceToHost);

    return particles;
}

extern "C" __declspec(dllexport) int cleanUp() {
    hipFree(dXPos);
    hipFree(dYPos);
    hipFree(dXVelo);
    hipFree(dYVelo);
    hipFree(dM);
    hipFree(dXPos2);
    hipFree(dYPos2);
    hipFree(dXVelo2);
    hipFree(dYVelo2);
    return 1;
}