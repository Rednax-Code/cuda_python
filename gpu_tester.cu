#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

// Kernel for adding two arrays
__global__ void arrayTestKernel(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate the global thread index
    if (idx < n) { // Ensure we don't access out-of-bounds memory
        int sum = 0;
        for (int i = 0; i < n; i++) {
            sum = sum + a[i];
        }
        c[idx] = a[idx] * b[idx] + sum / n;
    }
}

struct cIntArray {
    int* data;
    int length;
};

struct cFloatArray {
    double* data;
    double length;
};

extern "C" __declspec(dllexport) void freeArray(cIntArray* ptr) {
    if (ptr) {
        delete[] ptr->data;  // Free the dynamically allocated array
        delete ptr;          // Free the struct itself
    }
}

extern "C" __declspec(dllexport) const char* arrayTestSignature() {
    return "((list[int], list[int]), (list[int]))";
}

// Host function to add arrays
extern "C" __declspec(dllexport) cIntArray* arrayTest(cIntArray a, cIntArray b) {
    int *d_a, *d_b, *d_c;
    int n = a.length;

    if (b.length != n) {
        return nullptr;
    }

    // Allocate memory for the result
    cIntArray *result = new cIntArray;
    result->data = new int[a.length];
    result->length = a.length;

    // Allocate device memory and perform computations...
    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));
    
    hipMemcpyAsync(d_a, a.data, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b, b.data, n * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    arrayTestKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    hipMemcpy(result->data, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free device pointers
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return result;
}